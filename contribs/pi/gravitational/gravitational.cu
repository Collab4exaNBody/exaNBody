#include "hip/hip_runtime.h"
/*
Licensed to the Apache Software Foundation (ASF) under one
or more contributor license agreements.  See the NOTICE file
distributed with this work for additional information
regarding copyright ownership.  The ASF licenses this file
to you under the Apache License, Version 2.0 (the
"License"); you may not use this file except in compliance
with the License.  You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing,
software distributed under the License is distributed on an
"AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
KIND, either express or implied.  See the License for the
specific language governing permissions and limitations
under the License.
*/

#include <onika/log.h>
#include <onika/scg/operator.h>
#include <onika/scg/operator_factory.h>
#include <onika/scg/operator_slot.h>
#include <onika/math/basic_types.h>
#include <onika/physics/units.h>

#include <exanb/core/config.h> // for MAX_PARTICLE_NEIGHBORS constant
#include <exanb/core/grid.h>
#include <exanb/core/domain.h>
#include <exanb/compute/compute_cell_particle_pairs.h>
#include <exanb/core/make_grid_variant_operator.h>
#include <exanb/particle_neighbors/chunk_neighbors.h> // for MAX_PARTICLE_NEIGHBORS constant

// this allows for parallel compilation of templated operator for each available field set
namespace md
{
  using namespace exanb;

  // Gravitational Parameters
  struct GravitationalParms
  {
    double G = 0.0;
  };

  ONIKA_HOST_DEVICE_FUNC inline void gravitational_compute_energy(const GravitationalParms& p, const PairPotentialMinimalParameters& p_pair, double r, double& e, double& de)
  {
    assert( r > 0. );
    const double inv_r = 1.0 / r;
    e = - p.G * p_pair.m_atom_a.m_mass * p_pair.m_atom_b.m_mass * inv_r;
    de =  p.G * p_pair.m_atom_a.m_mass * p_pair.m_atom_b.m_mass * inv_r * inv_r;
  }

  // interaction potential compute functor
  struct alignas(onika::memory::DEFAULT_ALIGNMENT) GravitationalForceFunctor
  {
    // potential function parameters
    const GravitationalParms m_params;

    // concrete type of computation buffer and particle container may vary,
    // we use templates here to adapat to various situations
    template<class ComputePairBufferT, class CellParticlesT> 
    inline void operator () (
      size_t n,                          // number of neighbor particles
      const ComputePairBufferT& buffer,  // neighbors buffer
      double& fx,                        // central particle's force/X reference
      double& fy,                        // central particle's force/Y reference
      double& fz,                        // central particle's force/Z reference
      CellParticlesT* cells              // arrays of all cells, in case we need to chase for additional particle informations
      ) const
    {
      // local energy and force contributions to the particle
      double _fx = 0.;
      double _fy = 0.;
      double _fz = 0.;

#     pragma omp simd reduction(+:_fx,_fy,_fz)
      for(size_t i=0;i<n;i++)
      {
        const double r = std::sqrt(buffer.d2[i]);
        double pair_e=0.0, pair_de=0.0;
        gravitational_compute_energy( m_params, r, pair_e, pair_de );
        const auto interaction_weight = buffer.nbh_data.get(i);
        pair_de *= interaction_weight / r;        
        _fx += pair_de * buffer.drx[i];  // force is energy derivative multiplied by rij vector, sum force contributions for all neighbor particles
        _fy += pair_de * buffer.dry[i];
        _fz += pair_de * buffer.drz[i];
      }
      fx += _fx;
      fy += _fy;
      fz += _fz;
    }

    // ComputeBuffer less computation without virial
    template<class CellParticlesT>
    ONIKA_HOST_DEVICE_FUNC inline void operator () (
        Vec3d dr
      , double d2
      , double& fx
      , double& fy
      , double& fz
      , CellParticlesT* cells
      , size_t neighbor_cell
      , size_t neighbor_particle
      , double interaction_weight ) const
    {
      const double r = sqrt(d2);
      double pair_e = 0.0 , pair_de = 0.0;
      gravitational_compute_energy( m_params, r, pair_e, pair_de );
      pair_de *= interaction_weight / r;        
      fx += pair_de * dr.x;
      fy += pair_de * dr.y;
      fz += pair_de * dr.z;
    }

  };

}

namespace exanb
{

  // specialize functor traits to allow Cuda execution space
  template<>
  struct ComputePairTraits< md::GravitationalForceFunctor >
  {
    static inline constexpr bool RequiresBlockSynchronousCall = false;
    static inline constexpr bool ComputeBufferCompatible = true;
    static inline constexpr bool BufferLessCompatible = true;
    static inline constexpr bool CudaCompatible = true;
  };

}

// Yaml conversion operators, allows to read potential parameters from config file
namespace YAML
{

  template<> struct convert< md::GravitationalParms >
  {
    static bool decode(const Node& node, md::GravitationalParms & v)
    {
      v = md::GravitationalParms {};
      if( !node.IsMap() ) { return false; }
      v.G = node["G"].as<onika::physics::Quantity>().convert();
      return true;
    }
  };

}

namespace md
{
  using namespace exanb;

  template<
    class GridT,
    class = AssertGridHasFields< GridT, field::_fx ,field::_fy ,field::_fz >
    >
  class GravitationalForce : public OperatorNode
  {
    // ========= I/O slots =======================
    ADD_SLOT( GravitationalParms         , config          , INPUT        , REQUIRED , DocString{"Lennard-Jones potential parameters"} );
    ADD_SLOT( double                    , rcut            , INPUT        , 0.0 , DocString{"Cutoff distance"} );
    ADD_SLOT( exanb::GridChunkNeighbors , chunk_neighbors , INPUT        , exanb::GridChunkNeighbors{} , DocString{"neighbor list"} );
    ADD_SLOT( bool                      , ghost           , INPUT        , false , DocString{"Enables computation in ghost cells"});
    ADD_SLOT( Domain                    , domain          , INPUT        , REQUIRED , DocString{"Simulation domain"});
    ADD_SLOT( double                    , rcut_max        , INPUT_OUTPUT , 0.0 , DocString{"Updated max rcut"});
    ADD_SLOT( GridT                     , grid            , INPUT_OUTPUT , DocString{"Local sub-domain particles grid"} );

    // shortcut to the Compute buffer used (and passed to functor) by compute_pair_singlemat
    using ComputeBuffer = ComputePairBuffer2<false,false>;
    static inline constexpr FieldSet< field::_fx ,field::_fy ,field::_fz > compute_field_set = {};

  public:
    // Operator execution
    inline void execute () override final
    {
      assert( chunk_neighbors->number_of_cells() == grid->number_of_cells() );

      *rcut_max = std::max( *rcut , *rcut_max );
      if( grid->number_of_cells() == 0 ) { return; }

      ComputePairOptionalLocks<false> cp_locks {};
      exanb::GridChunkNeighborsLightWeightIt<false> nbh_it{ *chunk_neighbors };
      auto force_buf = make_compute_pair_buffer<ComputeBuffer>();
      GravitationalForceFunctor force_op = { *config };

      LinearXForm cp_xform { domain->xform() };
      auto optional = make_compute_pair_optional_args( nbh_it, ComputePairNullWeightIterator{} , cp_xform, cp_locks );
      compute_cell_particle_pairs( *grid, *rcut, *ghost, optional, force_buf, force_op , compute_field_set , parallel_execution_context() );      
    }

  };

  template<class GridT> using GravitationalForceTmpl = GravitationalForce<GridT>;

  // === register factories ===  
  ONIKA_AUTORUN_INIT(gravitational_force)
  {  
    OperatorNodeFactory::instance()->register_factory( "gravitational_force" , make_grid_variant_operator< GravitationalForceTmpl > );
  }

}
